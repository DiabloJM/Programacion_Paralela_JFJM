#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>

#define ARRAY_SIZE_X 128
#define ARRAY_SIZE_Y 128
#define BANK_SIZE 32

__global__ void padArray(int* array) {
// Shared memory with padding
    __shared__ int sharedArray[ARRAY_SIZE_X + ARRAY_SIZE_Y / BANK_SIZE];

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int index = bid * blockDim.x + tid;

// Load data into shared memory with padding
    sharedArray[tid] = array[index];
    __syncthreads();

// Access all keys from the original bank 0 in one clock pulse
    int offset = tid / BANK_SIZE;
    int newIndex = tid + offset;

// Use the modified index for accessing the padded shared memory
    int result = sharedArray[newIndex];
    array[index] = result;

// Print the result for demonstration
    printf("Thread %d: Original Value: %d, Padded Value: %d\n", tid, array[index], result);
}

int main() {
    int array[ARRAY_SIZE_X * ARRAY_SIZE_Y];

// Initialize array values (you can replace this with your data)
    for (int i = 0; i < (ARRAY_SIZE_X * ARRAY_SIZE_Y); ++i) {
        array[i] = (rand() % 10) + 1;
    }

    int* d_array;

// Allocate device memory
    hipMalloc((void**)&d_array, (ARRAY_SIZE_X * ARRAY_SIZE_Y) * sizeof(int));

// Copy array from host to device
    hipMemcpy(d_array, array, (ARRAY_SIZE_X * ARRAY_SIZE_Y) * sizeof(int), hipMemcpyHostToDevice);

// Define block and grid dimensions
    dim3 blockDim(BANK_SIZE);
    dim3 gridDim(((ARRAY_SIZE_X * ARRAY_SIZE_Y) + blockDim.x - 1) / blockDim.x);

// Launch kernel
    padArray<<<gridDim, blockDim>>>(d_array);

// Synchronize device to ensure print statements are displayed
    hipDeviceSynchronize();

// Free allocated memory
    hipFree(d_array);

    return 0;
}
